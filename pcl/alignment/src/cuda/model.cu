#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/binary_search.h>

#include "model.h"
#include "impl/ppf_utils.hpp"
#include "impl/util.hpp"
#include "impl/parallel_hash_array.hpp"
#include "kernel.h"
#include "book.h"

struct high_32_bits : public thrust::unary_function<unsigned long,unsigned int>{
    __host__ __device__
    unsigned int operator()(unsigned long i) const {
        return (unsigned int) (i >> 32);
    }
};


struct low_32_bits : public thrust::unary_function<unsigned long,unsigned int>{
    __host__ __device__
    unsigned int operator()(unsigned long i) const {
        return (unsigned int) (i & (-1ul >> 32));
    }
};

struct float16 {
    float f[16];
};

Model::Model(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n){
    this->initPPFs(points, normals, n);
    this->search_array = ParallelHashArray<float4>(*(this->modelPPFs));
}

Model::~Model(){
    // TODO
}

void Model::ppf_lookup(Scene *scene){

    #ifdef DEBUG
        hipEvent_t start, stop;
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));
    #endif

    thrust::device_vector<std::size_t> *sceneIndices =
        this->search_array.GetIndices(*(scene->getHashKeys()));
    // Steps 1-3
    // launch voting kernel instance for each scene reference point
    unsigned int lastIndex, lastCount;
    thrust::device_vector<unsigned long> *votes_old = new thrust::device_vector<unsigned long>(scene->getModelPPFs()->size()*this->modelPoints->size(),0);

    // populates parallel arrays votes and vecs_old
    int blocks = std::min(((int)(scene->getHashKeys()->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    ppf_vote_kernel<<<blocks,BLOCK_SIZE>>>
        (RAW_PTR(scene->getHashKeys()), RAW_PTR(sceneIndices),
         RAW_PTR(this->search_array.GetHashkeys()), RAW_PTR(this->search_array.GetCounts()),
         RAW_PTR(this->search_array.GetFirstHashkeyIndices()),
         RAW_PTR(this->search_array.GetHashkeyToDataMap()),
         RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
         this->n, RAW_PTR(scene->getModelPoints()),
         RAW_PTR(scene->getModelNormals()), scene->numPoints(),
         RAW_PTR(votes_old),
         scene->getHashKeys()->size());

    thrust::sort(votes_old->begin(), votes_old->end());
    this->votes = new thrust::device_vector<unsigned long>();
    this->voteCounts = new thrust::device_vector<unsigned int>();
    histogram_destructive(*votes_old, *(this->votes), *(this->voteCounts));

    this->transformations = new thrust::device_vector<float>(this->votes->size()*16);
    /* DEBUG */
    fprintf(stderr, "votes_size: %d\n", this->votes->size());
    /* DEBUG */

    blocks = std::min(((int)(this->votes->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);


    trans_calc_kernel2<<<blocks,BLOCK_SIZE>>>
        (RAW_PTR(votes),
         RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
         RAW_PTR(scene->getModelPoints()), RAW_PTR(scene->getModelNormals()),
         RAW_PTR(this->transformations),
         this->votes->size());

    this->transformation_trans = new thrust::device_vector<float3>(this->votes->size());
    this->transformation_rots = new thrust::device_vector<float4>(this->votes->size());
    mat2transquat_kernel<<<blocks,BLOCK_SIZE>>>
        (RAW_PTR(this->transformations),
         RAW_PTR(this->transformation_trans),
         RAW_PTR(this->transformation_rots),
         this->votes->size());

    thrust::device_vector<unsigned int> *nonunique_trans_hash =
        new thrust::device_vector<unsigned int>(this->votes->size());
    thrust::device_vector<unsigned int> *adjacent_trans_hash =
        new thrust::device_vector<unsigned int>(27*this->votes->size());
    trans2idx_kernel<<<blocks,BLOCK_SIZE>>>
        (RAW_PTR(this->transformation_trans),
         RAW_PTR(nonunique_trans_hash),
         RAW_PTR(adjacent_trans_hash),
         this->votes->size());

    this->key2transMap = new thrust::device_vector<unsigned int>(this->votes->size());
    thrust::sequence(key2transMap->begin(), key2transMap->end());
    thrust::sort_by_key(nonunique_trans_hash->begin(),
                        nonunique_trans_hash->end(),
                        key2transMap->begin());

    this->trans_hash = new thrust::device_vector<unsigned int>(this->votes->size());
    this->transCount = new thrust::device_vector<unsigned int>(this->votes->size());
    histogram_destructive(*nonunique_trans_hash, *(this->trans_hash), *(this->transCount));
    delete nonunique_trans_hash;
    this->firstTransIndex = new thrust::device_vector<unsigned int>(this->votes->size());
    thrust::exclusive_scan(this->transCount->begin(),
                           this->transCount->end(),
                           this->firstTransIndex->begin());

    thrust::device_vector<unsigned int> *transIndices =
        new thrust::device_vector<unsigned int>(adjacent_trans_hash->size());
    thrust::lower_bound(this->trans_hash->begin(),
                        this->trans_hash->end(),
                        adjacent_trans_hash->begin(),
                        adjacent_trans_hash->end(),
                        transIndices->begin());

    // write_device_vector("adjacent_trans_hash", adjacent_trans_hash);
    // write_device_vector("transCount", transCount);

    this->vote_counts_out = new thrust::device_vector<unsigned int>(*(this->voteCounts));
    blocks = std::min(((int)(this->votes->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    rot_clustering_kernel<<<blocks,BLOCK_SIZE>>>
        (RAW_PTR(this->transformation_trans), RAW_PTR(this->transformation_rots),
         RAW_PTR(this->voteCounts), RAW_PTR(adjacent_trans_hash),
         RAW_PTR(transIndices), RAW_PTR(this->trans_hash),
         RAW_PTR(this->transCount), RAW_PTR(this->firstTransIndex),
         RAW_PTR(this->key2transMap), RAW_PTR(this->vote_counts_out),
         this->votes->size());

    thrust::sort_by_key(this->vote_counts_out->begin(),
                        this->vote_counts_out->end(),
                        thrust::device_ptr<struct float16>((struct float16 *) RAW_PTR(this->transformations)),
                        thrust::greater<unsigned int>());


    // thrust::device_vector<unsigned int> *uniqueSceneRefPts =
    //     new thrust::device_vector<unsigned int>(this->votes->size());
    // this->maxval = new thrust::device_vector<unsigned int>(this->votes->size());
    // thrust::device_vector<unsigned int> *maxModelAngleCode =
    //     new thrust::device_vector<unsigned int>(this->votes->size());

    // thrust::reduce_by_key
    //     (// key input: step function that increments for every row
    //      thrust::make_transform_iterator(votes->begin()+1, high_32_bits()),
    //      thrust::make_transform_iterator(votes->end(), high_32_bits()),
    //      // value input: (value, index) tuple
    //      thrust::make_zip_iterator(thrust::make_tuple(voteCounts->begin()+1,
    //                                                   thrust::make_transform_iterator(votes->begin()+1,
    //                                                                                   low_32_bits()))),
    //      uniqueSceneRefPts->begin(),
    //      thrust::make_zip_iterator(thrust::make_tuple(this->maxval->begin(),
    //                                                   maxModelAngleCode->begin())),
    //      thrust::equal_to<unsigned int>(),
    //      // compare by first element of tuple
    //      thrust::maximum<thrust::tuple<unsigned int, unsigned int> >());



    // Step 8, 9
    // call trans_calc_kernel
    // this->transformations = new thrust::device_vector<float>(this->votes->size()*16);

    // blocks = std::min(((int)(this->votes->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    // trans_calc_kernel<<<blocks,BLOCK_SIZE>>>
    //     (RAW_PTR(uniqueSceneRefPts), RAW_PTR(maxModelAngleCode),
    //      RAW_PTR(this->modelPoints), RAW_PTR(this->modelNormals),
    //      RAW_PTR(scene->getModelPoints()), RAW_PTR(scene->getModelNormals()),
    //      RAW_PTR(this->transformations),
    //      this->votes->size());

    #ifdef DEBUG
        // end cuda timer
        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
        fprintf(stderr, "Time to lookup model:  %3.1f ms\n", elapsedTime);
    #endif
}

void Model::accumulateVotes(){
    this->voteCodes = new thrust::device_vector<unsigned long>();
    this->voteCounts = new thrust::device_vector<unsigned int>();
    histogram(*(this->votes), *(this->voteCodes), *(this->voteCounts));
}

thrust::device_vector<float>* Model::getTransformations(){
    return this->transformations;
}
