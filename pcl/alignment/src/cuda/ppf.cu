#include "hip/hip_runtime.h"
#include "ppf.h"

#include <sys/types.h>
#include <sys/stat.h>

#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include <boost/format.hpp>
#include <boost/log/trivial.hpp>
#include <hip/hip_runtime.h>
#include <Eigen/Core>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <hip/hip_vector_types.h>

#include "impl/util.hpp"
#include "kernel.h"
#include "model.h"


std::vector<std::vector<Eigen::Matrix4f>> ppf_registration(
    std::vector<pcl::PointCloud<pcl::PointNormal>::Ptr> scene_clouds,
    std::vector<pcl::PointCloud<pcl::PointNormal>::Ptr> model_clouds,
    std::vector<float> model_d_dists, unsigned int ref_point_downsample_factor,
    float vote_count_threshold, bool cpu_clustering,
    bool use_l1_norm, bool use_averaged_clusters,
    int devUse, float *model_weights){

    int numDevices;
    HANDLE_ERROR(hipGetDeviceCount(&numDevices));
    BOOST_LOG_TRIVIAL(info) << boost::format("Found %d CUDA devices:") % numDevices;
    hipDeviceProp_t prop;
    for(int i = 0; i < numDevices; i++){
        hipGetDeviceProperties(&prop, i);
        BOOST_LOG_TRIVIAL(info) << boost::format("%d) name: %s") % i % prop.name;
    }
    HANDLE_ERROR(hipSetDevice(std::min(numDevices-1, devUse)));
    int devNum;
    HANDLE_ERROR(hipGetDevice(&devNum));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, devNum));
    BOOST_LOG_TRIVIAL(info) << boost::format("Using device %d, %s") % devNum % prop.name;

    // cuda setup
    int blocks = prop.multiProcessorCount;
    BOOST_LOG_TRIVIAL(debug) << boost::format("multiProcessorCount: %d") % blocks;

    std::vector<std::vector<Eigen::Matrix4f>> results;

    for(int i = 0; i < scene_clouds.size(); i++){
      pcl::PointCloud<pcl::PointNormal>::Ptr scene_cloud = scene_clouds[i];
        // build model description

        results.push_back(std::vector<Eigen::Matrix4f>());

        for(int j = 0; j < model_clouds.size(); j++){
            // The d_dist for the scene must match the d_dist for the model, so
            // we need to re-compute (or at least re-downsample, which is about
            // as expensive) the scene PPFs fpr each model.
            Scene *scene = new Scene(scene_cloud.get(), model_d_dists[j], ref_point_downsample_factor);
            pcl::PointCloud<pcl::PointNormal>::Ptr model_cloud = model_clouds[j];
            Model *model = new Model(model_cloud.get(), model_d_dists[j], vote_count_threshold,
                                     cpu_clustering, use_l1_norm, use_averaged_clusters);

            model->ppf_lookup(scene);

            Eigen::Matrix4f T;
            if(cpu_clustering){
                T = model->cpu_transformations[0].pose.matrix();
            }
            else{
                // TODO: copy only the first transformations instead of the entire vector.
                thrust::host_vector<float> transformations =
                    thrust::host_vector<float>(model->getTransformations());
                for(int r = 0; r < 4; r++){
                    for(int c = 0; c < 4; c++){
                        T(r,c) = transformations[model->max_idx*16 + r*4+c];
                    }
                }
                thrust::host_vector<float3> transformation_trans(*model->transformation_trans);
                thrust::host_vector<float4> transformation_rots(*model->transformation_rots);
                // quat2hrotmat(transformation_rots[model->max_idx], (float (*)[4]) transformations.data());
                T(0, 3) = transformation_trans[model->max_idx].x;
                T(1, 3) = transformation_trans[model->max_idx].y;
                T(2, 3) = transformation_trans[model->max_idx].z;
            }

            BOOST_LOG_TRIVIAL(info) << "Found transformation:\n" << T;
            results.back().push_back(T);
            delete model;
            delete scene;
        }
    }


    hipDeviceReset();

    return results;
}
