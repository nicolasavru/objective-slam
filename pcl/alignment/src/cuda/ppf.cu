#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <sys/types.h>
#include <sys/stat.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "kernel.h"
#include "book.h"
#include "model.h"

using namespace std;

#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>

#include <Eigen/Core>

void test_histogram(char *point_path, int N){
    FILE *points_fin;
    size_t result1;

    points_fin = fopen(point_path, "rb");
    if(points_fin==NULL){fputs ("File error: point_fin",stderr); exit (1);}

    thrust::host_vector<unsigned int> *points = new thrust::host_vector<unsigned int>(N);

    if (points == NULL) {fputs ("Memory error: points",stderr); exit (2);}

    result1 = fread(RAW_PTR(points),sizeof(unsigned int),N,points_fin);
    thrust::device_vector<unsigned int> *d_points = new thrust::device_vector<unsigned int>(*points);

    thrust::sort(d_points->begin(), d_points->end());

    // create histogram of hash keys
    // https://code.google.com/p/thrust/source/browse/examples/histogram.cu
    unsigned int num_bins = thrust::inner_product(d_points->begin(), d_points->end() - 1,
                                                  d_points->begin() + 1,
                                                  (unsigned int) 1,
                                                  thrust::plus<unsigned int>(),
                                                  thrust::not_equal_to<unsigned int>());

    /* DEBUG */
    fprintf(stderr, "num_bins: %d\n", num_bins);
    /* DEBUG */

    thrust::device_vector<unsigned int> *hashKeys = new thrust::device_vector<unsigned int>(num_bins);
    thrust::device_vector<unsigned int> *ppfCount = new thrust::device_vector<unsigned int>(num_bins);

    thrust::reduce_by_key(d_points->begin(), d_points->end(),
                          thrust::constant_iterator<unsigned int>(1),
                          hashKeys->begin(),
                          ppfCount->begin());


    thrust::host_vector<unsigned int> *A = new thrust::host_vector<unsigned int>(*ppfCount);

    for (int i = 0; i < num_bins; i++){
        fprintf(stderr, "%u: %u %u\n", i, (*A)[i]);
    }
}

void ptr_test_cu(pcl::PointCloud<pcl::PointNormal> *scene_cloud_ptr){
    fprintf(stderr, "foo-1: %p, %lu, %lu\n", scene_cloud_ptr, scene_cloud_ptr->points.size(), scene_cloud_ptr->size());
}

void ptr_test_cu2(pcl::PointCloud<pcl::PointNormal> scene_cloud){
    fprintf(stderr, "foo-2: %lu, %lu\n", scene_cloud.points.size(), scene_cloud.size());
}

void ptr_test_cu3(pcl::PointCloud<pcl::PointNormal> &scene_cloud){
    fprintf(stderr, "foo-3: %lu, %lu\n", scene_cloud.points.size(), scene_cloud.size());
}

void ptr_test_cu4(const pcl::PointCloud<pcl::PointNormal> &scene_cloud){
    fprintf(stderr, "foo-4: %lu, %lu\n", scene_cloud.points.size(), scene_cloud.size());
}


std::vector<std::vector<Eigen::Matrix4f>> ppf_registration(
    std::vector<pcl::PointCloud<pcl::PointNormal>::Ptr> scene_clouds,
    std::vector<pcl::PointCloud<pcl::PointNormal>::Ptr> model_clouds,
    std::vector<pcl::PointCloud<pcl::PointNormal>::Ptr> empty_clouds,
    std::vector<float> model_d_dists, unsigned int ref_point_downsample_factor,
    int devUse, float *model_weights){
    int *device_array = 0;
    // HANDLE_ERROR(hipMalloc((void**)&device_array, 1024*sizeof(int)));

    int numDevices;
    HANDLE_ERROR(hipGetDeviceCount(&numDevices));
    fprintf(stderr, "numDevices: %d\n", numDevices);
    hipDeviceProp_t prop;
    for(int i = 0; i < numDevices; i++){
        hipGetDeviceProperties(&prop, i);
        fprintf(stderr, "%d) name: %s\n", i, prop.name);
    }
    // HANDLE_ERROR(hipSetDevice(devUse));
    HANDLE_ERROR(hipSetDevice(std::min(numDevices-1, devUse)));
    int devNum;
    HANDLE_ERROR(hipGetDevice(&devNum));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, devNum));
    fprintf(stderr, "Using device %d, %s: \n", devNum, prop.name);
    // thrust::device_vector<float3> foo(1024);

    // cuda setup
    int blocks = prop.multiProcessorCount;
    /* DEBUG */
    fprintf(stderr, "blocks_multiproccount: %d\n", blocks);
    /* DEBUG */

    std::vector<std::vector<Eigen::Matrix4f>> results;

    for(int i = 0; i < scene_clouds.size(); i++){
      pcl::PointCloud<pcl::PointNormal>::Ptr scene_cloud = scene_clouds[i];
        // build model description
        // pcl::PointCloud<pcl::PointNormal> *scene_cloud_ptr = scene_clouds[0].get();
        // pcl::PointCloud<pcl::PointNormal> *object_cloud_ptr = model_clouds[0].get();

        results.push_back(std::vector<Eigen::Matrix4f>());

        for(int j = 0; j < model_clouds.size(); j++){
            // The d_dist for the scene must match the d_dist for the model, so
            // we need to re-compute (or at least re-downsample, which is about
            // as expensive) the scene PPFs fpr each model.
            Scene *scene = new Scene(scene_cloud.get(), model_d_dists[j], ref_point_downsample_factor);
            pcl::PointCloud<pcl::PointNormal>::Ptr model_cloud = model_clouds[j];
            Model *model = new Model(model_cloud.get(), model_d_dists[j]);
            Eigen::Matrix4f T;

            // thrust::host_vector<float> optimal_weights(model->OptimizeWeights(empty_clouds, 4));
            // model->modelPointVoteWeights = thrust::device_vector<float>(optimal_weights);
            // for(int i = 0; i < object_cloud_ptr->size(); i++){
            //     model_weights[i] = optimal_weights[i];
            // }
            model->ppf_lookup(scene);

            // copy ppfs back to host
            // TODO: copy only the first transformations instead of the entire vector.
            thrust::host_vector<float> transformations =
                thrust::host_vector<float>(model->getTransformations());
            // thrust::host_vector<unsigned int> *maxval = new thrust::host_vector<unsigned int>(*model->maxval);
            // thrust::host_vector<float> *maxval =
            //     new thrust::host_vector<float>(*model->vote_counts_out);

            // write out transformations
            // (*maxval)[0] is all the unallocated votes
            // float threshold = 0.8 * (*maxval)[1];
            // /* DEBUG */
            // fprintf(stderr, "threshold: %f\n", threshold);
            // /* DEBUG */
            // for (int i=1; (*maxval)[i] > threshold; i++){
            //    cout << "num_votes: " << (*maxval)[i] << endl;
            //    cout << "transforms(:,:," << i << ") = [";
            //    for (int j=0; j<4; j++){
            //        for (int k=0; k<4; k++){
            //            cout << transformations[i*16+j*4+k] << " ";
            //        }
            //        cout << ";" << endl;
            //    }
            //    cout << "];" << endl;
            //    cout << endl << endl;
            // }

            for (int j=0; j<4; j++){
                for (int k=0; k<4; k++){
                    // T(j,k) = transformations[16+j*4+k];
                    T(j,k) = transformations[j*4+k];
                }
            }
            cout << T << endl;
            results.back().push_back(T);
            delete model;
            delete scene;
        }
    }


    hipDeviceReset();

    return results;
}
