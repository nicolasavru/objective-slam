#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <pcl/point_types.h>
#include <pcl/point_cloud.h>
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>

#include "scene.h"
#include "kernel.h"
#include "book.h"
#include "impl/util.hpp"

Scene::Scene(){}

Scene::Scene(pcl::PointCloud<pcl::PointNormal> *cloud_ptr, float d_dist){
    /* DEBUG */
    fprintf(stderr, "foo0: %d\n", cloud_ptr->size());
    /* DEBUG */
    this->cloud_ptr = cloud_ptr;
    thrust::host_vector<float3> *points =
        new thrust::host_vector<float3>(cloud_ptr->size());
    thrust::host_vector<float3> *normals =
        new thrust::host_vector<float3>(cloud_ptr->size());

    for(int i = 0; i < cloud_ptr->size(); i++){
        (*points)[i].x = (*cloud_ptr)[i].x;
        (*points)[i].y = (*cloud_ptr)[i].y;
        (*points)[i].z = (*cloud_ptr)[i].z;
        (*normals)[i].x = (*cloud_ptr)[i].normal_x;
        (*normals)[i].y = (*cloud_ptr)[i].normal_y;
        (*normals)[i].z = (*cloud_ptr)[i].normal_z;
    }

    this->d_dist = d_dist;

    this->initPPFs(points, normals, cloud_ptr->size(), d_dist, 1);
    // thrust::host_vector<float3> *host_scene_modelnormals =
    //     new thrust::host_vector<float3>(*this->modelNormals);
    // for(int i = 0; i < host_scene_modelnormals->size(); i++){
    //     /* DEBUG */
    //     fprintf(stdout, "host_scene_modelnormals[%u]: %f, %f, %f\n", i,
    //             (*host_scene_modelnormals)[i].x, (*host_scene_modelnormals)[i].y, (*host_scene_modelnormals)[i].z);
    //     /* DEBUG */
    // }
    // thrust::host_vector<float4> *host_scene_modelppfs =
    //     new thrust::host_vector<float4>(*this->modelPPFs);
    // for(int i = 0; i < host_scene_modelppfs->size(); i++){
    //     /* DEBUG */
    //     fprintf(stdout, "host_scene_modelppfs[%u]: %f, %f, %f, %f\n", i,
    //             (*host_scene_modelppfs)[i].x, (*host_scene_modelppfs)[i].y, (*host_scene_modelppfs)[i].z,
    //             (*host_scene_modelppfs)[i].w);
    //     /* DEBUG */
    // }

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    this->hashKeys = new thrust::device_vector<unsigned int>(this->modelPPFs->size());

    int blocks = std::min(((int)(this->modelPPFs->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    ppf_hash_kernel<<<blocks,BLOCK_SIZE>>>(RAW_PTR(this->modelPPFs),
                                           RAW_PTR(this->hashKeys),
                                           this->modelPPFs->size());

    // thrust::host_vector<std::size_t> *host_scene_hashkeys_init =
    //     new thrust::host_vector<std::size_t>(*this->hashKeys);
    // for(int i = 0; i < host_scene_hashkeys_init->size(); i++){
    //     /* DEBUG */
    //     fprintf(stdout, "host_scene_hashkeys_init[%u]: %u\n", i, (*host_scene_hashkeys_init)[i]);
    //     /* DEBUG */
    // }

}

Scene::~Scene(){
    delete this->modelPoints;
    delete this->modelNormals;
    delete this->modelPPFs;
    // delete this->hashKeys;
}

void Scene::initPPFs(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n,
                     float d_dist, int ref_point_downsample_factor){
    this->n = n;
    // check if these are used later or can be discarded after this function
    this->modelPoints = new thrust::device_vector<float3>(*points);
    this->modelNormals = new thrust::device_vector<float3>(*normals);
    this->modelPPFs = new thrust::device_vector<float4>(n*n);

#ifdef DEBUG
        fprintf(stderr, "n: %d\n", n);

        // start cuda timer
        hipEvent_t start, stop;
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));
    #endif

    // This will crash if n = 0;
    int blocks = std::min(((int)(this->n + BLOCK_SIZE) - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    // MATLAB drost.m:59, all of model_description.m
    // ppf_kernel computes ppfs and descritizes them, but does *not* hash them
    // hashing is done by ppf_hash_kernel, called only for model, not scene (model.cu:46)
    ppf_kernel<<<blocks,BLOCK_SIZE>>>(RAW_PTR(this->modelPoints),
                                      RAW_PTR(this->modelNormals),
                                      RAW_PTR(this->modelPPFs),
                                      n, ref_point_downsample_factor, this->d_dist);

    #ifdef DEBUG
        // end cuda timer
        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
        fprintf(stderr, "Time to generate PPFs: %3.1f ms\n", elapsedTime);
    #endif
}

int Scene::numPoints(){
    return this->n;
}

thrust::device_vector<float3> *Scene::getModelPoints(){
    return this->modelPoints;
}

thrust::device_vector<float3> *Scene::getModelNormals(){
    return this->modelNormals;
}
thrust::device_vector<float4> *Scene::getModelPPFs(){
    return this->modelPPFs;
}

thrust::device_vector<unsigned int>* Scene::getHashKeys(){
    return this->hashKeys;
}
