#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>                // Stops underlining of __global__
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>

#include "scene.h"
#include "kernel.h"
#include "book.h"
#include "impl/util.hpp"

Scene::Scene(){}

Scene::Scene(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n){
    this->initPPFs(points, normals, n);
    this->hashKeys = new thrust::device_vector<unsigned int>(this->modelPPFs->size());

    int blocks = std::min(((int)(this->modelPPFs->size()) + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    ppf_hash_kernel<<<blocks,BLOCK_SIZE>>>(RAW_PTR(this->modelPPFs),
                                           RAW_PTR(this->hashKeys),
                                           this->modelPPFs->size());
}

Scene::~Scene(){
    delete this->modelPoints;
    delete this->modelNormals;
    delete this->modelPPFs;
    delete this->hashKeys;
}

void Scene::initPPFs(thrust::host_vector<float3> *points, thrust::host_vector<float3> *normals, int n){
    this->n = n;
    // check if these are used later or can be discarded after this function
    this->modelPoints = new thrust::device_vector<float3>(*points);
    this->modelNormals = new thrust::device_vector<float3>(*normals);
    this->modelPPFs = new thrust::device_vector<float4>(n*n);

    #ifdef DEBUG
        fprintf(stderr, "n: %d\n", n);

        // start cuda timer
        hipEvent_t start, stop;
        HANDLE_ERROR(hipEventCreate(&start));
        HANDLE_ERROR(hipEventCreate(&stop));
        HANDLE_ERROR(hipEventRecord(start, 0));
    #endif

    int blocks = std::min(((int)(this->n + BLOCK_SIZE) - 1) / BLOCK_SIZE, MAX_NBLOCKS);
    // MATLAB drost.m:59, all of model_description.m
    // ppf_kernel computes ppfs and descritizes them, but does *not* hash them
    // hashing is done by ppf_hash_kernel, called only for model, not scene (model.cu:46)
    ppf_kernel<<<blocks,BLOCK_SIZE>>>(RAW_PTR(this->modelPoints),
                                      RAW_PTR(this->modelNormals),
                                      RAW_PTR(this->modelPPFs),
                                      n);

    #ifdef DEBUG
        // end cuda timer
        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));
        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
        fprintf(stderr, "Time to generate PPFs: %3.1f ms\n", elapsedTime);
    #endif
}

int Scene::numPoints(){
    return this->n;
}

thrust::device_vector<float3> *Scene::getModelPoints(){
    return this->modelPoints;
}

thrust::device_vector<float3> *Scene::getModelNormals(){
    return this->modelNormals;
}
thrust::device_vector<float4> *Scene::getModelPPFs(){
    return this->modelPPFs;
}

thrust::device_vector<unsigned int>* Scene::getHashKeys(){
    return this->hashKeys;
}
